#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define BLOCK_DIM 8
// const int NUM_REPS = 100;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code == hipSuccess) return;
    fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

double timeStamp() {
    struct timeval tv; 
    gettimeofday(&tv, NULL);
    return tv.tv_usec / 1000.0 + tv.tv_sec;
}

void displayResults(float *A, float *T, int M, int N){
    // display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < M; ++i) {
		for (int j = 0; j < N; ++j) {
			printf("A: %f ", A[i * N + j]);
		}
		printf("\n");
	}

	printf("----------\n");
	printf("Transpose: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("%f ", T[i * M + j]);
		}
		printf("\n");
	}
}

void transposeCPU(float *A, float *T, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            T[j * M + i] = A[i * N + j];
        }
    }
}

void validateResults(float *h_A, float *h_T, int M, int N){
    // Allocate memory for the transpose matrix on CPU
    float *h_T_CPU = (float *)malloc(M * N * sizeof(float));
    // Transpose matrix A on CPU
    transposeCPU(h_A, h_T_CPU, M, N);

    // Validate the results
    int incorrectCount = 0;
    for (int i = 0; i < M * N; ++i) {
        if (abs(h_T_CPU[i] - h_T[i]) > 1e-5) {
            incorrectCount++;
            // Uncomment the next line to print each incorrect element
            // printf("Mismatch at index %d, CPU: %f, GPU: %f\n", i, h_T_CPU[i], h_T[i]);
        }
    }
    
    if (incorrectCount == 0) {
        printf("Validation Passed!\n");
    } else {
        printf("Validation Failed: %d elements incorrect.\n", incorrectCount);
    }

    // Clean up CPU transpose matrix
    free(h_T_CPU);
}

__global__ void transposeNaive(float *d_A, float *d_T, int M, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// swap elements via transpose
	if (row < M && col < N) {
		d_T[col * M + row] = d_A[row * N + col];
	}
}

__global__ void transpose(float *d_A, float *d_T, int M, int N)
{
	__shared__ float block[TILE_DIM][TILE_DIM+1];
	
	unsigned int row = blockIdx.y * TILE_DIM + threadIdx.y;
	unsigned int col = blockIdx.x * TILE_DIM + threadIdx.x;
    unsigned int index_in = row * N + col;
    unsigned int index_out = col * M + row;
	
    if((row < M) && (col < N) && (index_in < M*N)) {
		block[threadIdx.y][threadIdx.x] = d_A[index_in];
	}

	__syncthreads();

	row = blockIdx.y * TILE_DIM + threadIdx.x;
	col = blockIdx.x * TILE_DIM + threadIdx.y;
	if((row < M) && (col < N) && (index_out < M*N)) {
		d_T[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

torch::Tensor forward(torch::Tensor A) {
    // A and B are 4D tensors in row major format: 
    // A = (batchsize, head, M, K)
    const int M = A.size(2);
    const int N = A.size(3);

    // Initialize A, Z to host memory
    torch::Tensor C = torch::zeros({A.size(0), A.size(1), N, M}, A.options().device(torch::kCUDA));
    auto A_data = A.data_ptr<float>();
    auto C_data = C.data_ptr<float>();

	dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
	dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y - 1)/blockDim.y);

    double start, end;
    start = timeStamp();
    transpose<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    // transposeNaive<<<gridDim, blockDim>>>(A_data, C_data, M, N);
    hipDeviceSynchronize();
    end = timeStamp();

    printf("GPU execution time: %.4f milliseconds\n", (end-start));

	return C;
}


// % nvcc -arch sm_89 transpose.cu -o transpose
// % transpose