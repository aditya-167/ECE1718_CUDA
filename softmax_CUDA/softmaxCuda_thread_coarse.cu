#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <chrono>

#define THREADS_PER_BLOCK 256

// CUDA kernel for softmax calculation with thread coalescing and coarsening
__global__ void softmax_kernel_coalesced_coarsened(float *input, float *output, int rows, int cols, int coarsening_factor) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < rows) {
        float max_val = input[idx * cols];
        for (int i = 1; i < cols; i++) {
            float val = input[idx * cols + i];
            max_val = (val > max_val) ? val : max_val;
        }

        float sum_exp = 0.0f;
        // Thread coarsening: each thread handles multiple elements
        for (int i = 0; i < cols; i += coarsening_factor) {
            float exp_sum = 0.0f;
            // Compute the sum of exponentials for the coarsened group
            for (int j = 0; j < coarsening_factor && i + j < cols; j++) {
                float exp_val = expf(input[idx * cols + i + j] - max_val);
                output[idx * cols + i + j] = exp_val;
                exp_sum += exp_val;
            }
            // Accumulate the sum of exponentials for normalization
            sum_exp += exp_sum;
        }
        // Normalize the softmax values
        for (int i = 0; i < cols; i += coarsening_factor) {
            for (int j = 0; j < coarsening_factor && i + j < cols; j++) {
                output[idx * cols + i + j] /= sum_exp;
            }
        }
    }
}

// Host function to calculate softmax
void softmax_cpu(float *input, float *output, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        float max_val = input[i * cols];
        for (int j = 1; j < cols; j++) {
            float val = input[i * cols + j];
            max_val = (val > max_val) ? val : max_val;
        }

        float sum_exp = 0.0f;
        for (int j = 0; j < cols; j++) {
            float exp_val = expf(input[i * cols + j] - max_val);
            output[i * cols + j] = exp_val;
            sum_exp += exp_val;
        }

        for (int j = 0; j < cols; j++) {
            output[i * cols + j] /= sum_exp;
        }
    }
}

// Function to verify GPU results against CPU results
bool verify_result(float *gpu_result, float *cpu_result, int size) {
    for (int i = 0; i < size; i++) {
        if (fabs(gpu_result[i] - cpu_result[i]) > 1e-5) {
            std::cout << "Verification failed at index " << i << ": GPU - " << gpu_result[i] << ", CPU - " << cpu_result[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int rows = 4096;
    const int cols = 4096;
    const int size = rows * cols * sizeof(float);
    const int coarsening_factor = 4; // You can adjust this value as needed

    // Allocate memory on the host
    float *input_host = new float[size];
    float *output_host_cpu = new float[size];
    float *output_host_gpu = new float[size];

    // Initialize input data on the host
    srand(time(NULL));
    for (int i = 0; i < rows * cols; i++) {
        input_host[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 10.0f;
    }

    // Allocate memory on the device
    float *input_device, *output_device;
    hipMalloc((void**)&input_device, size);
    hipMalloc((void**)&output_device, size);

    // Copy input data from host to device
    hipMemcpy(input_device, input_host, size, hipMemcpyHostToDevice);

    // Create CUDA events for timing GPU execution
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // Record start event for GPU
    hipEventRecord(start_gpu);

    // Launch GPU kernel
    int num_blocks = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    softmax_kernel_coalesced_coarsened<<<num_blocks, THREADS_PER_BLOCK>>>(input_device, output_device, rows, cols, coarsening_factor);
    hipDeviceSynchronize();

    // Record stop event for GPU
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    // Calculate GPU execution time
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    // Copy output data from device to host
    hipMemcpy(output_host_gpu, output_device, size, hipMemcpyDeviceToHost);

    // Perform softmax calculation on the CPU and measure time
    auto start_cpu = std::chrono::high_resolution_clock::now();
    softmax_cpu(input_host, output_host_cpu, rows, cols);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    auto duration_cpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu);

    // Verify GPU results against CPU results
    bool result = verify_result(output_host_gpu, output_host_cpu, rows * cols);
    if (result) {
        std::cout << "GPU computation matches CPU computation." << std::endl;
    } else {
        std::cout << "GPU computation does not match CPU computation." << std::endl;
    }

    // Print GPU execution time
    std::cout << "GPU Execution Time: " << gpu_time << " milliseconds" << std::endl;

    // Print CPU execution time
    std::cout << "CPU Execution Time: " << duration_cpu.count() << " milliseconds" << std::endl;

    // Free memory
    delete[] input_host;
    delete[] output_host_cpu;
    delete[] output_host_gpu;
    hipFree(input_device);
    hipFree(output_device);

    // Destroy events
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    return 0;
}

/*
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <cuda_runtime.h>

#define THREADS_PER_BLOCK 256

// CUDA kernel for softmax calculation with vectorization
__global__ void softmax_kernel_vectorized(float *input, float *output, int rows, int cols) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < rows) {
        // Initialize max_val and sum_exp using vectorized instructions
        float max_val = input[idx * cols];
        float sum_exp = 0.0f;

        // Compute max_val and sum_exp using vectorized instructions
        for (int i = 1; i < cols; i += 4) {
            float4 input_vec = reinterpret_cast<float4*>(input + idx * cols)[i / 4];
            max_val = fmaxf(max_val, fmaxf(fmaxf(input_vec.x, input_vec.y), fmaxf(input_vec.z, input_vec.w)));
            sum_exp += expf(input_vec.x - max_val) + expf(input_vec.y - max_val) + expf(input_vec.z - max_val) + expf(input_vec.w - max_val);
        }

        // Compute softmax values using vectorized instructions
        for (int i = 0; i < cols; i += 4) {
            float4 input_vec = reinterpret_cast<float4*>(input + idx * cols)[i / 4];
            float4 exp_val = make_float4(expf(input_vec.x - max_val), expf(input_vec.y - max_val), expf(input_vec.z - max_val), expf(input_vec.w - max_val));
            sum_exp = exp_val.x + exp_val.y + exp_val.z + exp_val.w;
            float4 softmax_val = make_float4(exp_val.x / sum_exp, exp_val.y / sum_exp, exp_val.z / sum_exp, exp_val.w / sum_exp);
            reinterpret_cast<float4*>(output + idx * cols)[i / 4] = softmax_val;
        }
    }
}

// Host function to calculate softmax on CPU
void softmax_cpu(float *input, float *output, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        // Compute max value
        float max_val = input[i * cols];
        for (int j = 1; j < cols; j++) {
            max_val = fmaxf(max_val, input[i * cols + j]);
        }

        // Compute exponentials and sum
        float sum_exp = 0.0f;
        for (int j = 0; j < cols; j++) {
            float exp_val = expf(input[i * cols + j] - max_val);
            output[i * cols + j] = exp_val;
            sum_exp += exp_val;
        }

        // Normalize softmax values
        for (int j = 0; j < cols; j++) {
            output[i * cols + j] /= sum_exp;
        }
    }
}

// Function to verify GPU results against CPU results
bool verify_result(float *gpu_result, float *cpu_result, int size) {
    for (int i = 0; i < size; i++) {
        if (fabs(gpu_result[i] - cpu_result[i]) > 1e-5) {
            std::cout << "Verification failed at index " << i << ": GPU - " << gpu_result[i] << ", CPU - " << cpu_result[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int rows = 10000; // Increase rows and cols for larger matrix
    const int cols = 1000;
    const int size = rows * cols * sizeof(float);

    // Allocate memory on the host
    float *input_host = new float[size];
    float *output_host_cpu = new float[size];
    float *output_host_gpu = new float[size];

    // Initialize input data on the host
    srand(time(NULL));
    for (int i = 0; i < rows * cols; i++) {
        input_host[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 10.0f;
    }

    // Allocate memory on the device
    float *input_device, *output_device;
    cudaMalloc((void**)&input_device, size);
    cudaMalloc((void**)&output_device, size);

    // Copy input data from host to device
    cudaMemcpy(input_device, input_host, size, cudaMemcpyHostToDevice);

    // Launch GPU kernel
    int num_blocks = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    softmax_kernel_vectorized<<<num_blocks, THREADS_PER_BLOCK>>>(input_device, output_device, rows, cols);
    cudaDeviceSynchronize();

    // Copy output data from device to host
    cudaMemcpy(output_host_gpu, output_device, size, cudaMemcpyDeviceToHost);

    // Perform softmax calculation on the CPU
    softmax_cpu(input_host, output_host_cpu, rows, cols);

    // Verify GPU results against CPU results
    bool result = verify_result(output_host_gpu, output_host_cpu, rows * cols);
    if (result) {
        std::cout << "GPU computation matches CPU computation." << std::endl;
    } else {
        std::cout << "GPU computation does not match CPU computation." << std::endl;
    }

    // Free memory
    delete[] input_host;
    delete[] output_host_cpu;
    delete[] output_host_gpu;
    cudaFree(input_device);
    cudaFree(output_device);

    return 0;
}
*/